#include "hip/hip_runtime.h"
#include <torch/extension.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

namespace
{
    template <typename scalar_t>
    __global__ void vadd_cuda_forward_kernel(
        const torch::PackedTensorAccessor<scalar_t, 1, torch::RestrictPtrTraits, size_t> A,
        const torch::PackedTensorAccessor<scalar_t, 1, torch::RestrictPtrTraits, size_t> B,
        torch::PackedTensorAccessor<scalar_t, 1, torch::RestrictPtrTraits, size_t> C)
    {
        const int i = blockDim.x * blockIdx.x + threadIdx.x;
        if (i < A.size(0))
        {
            C[i] = A[i] + B[i];
        }
    }
}

torch::Tensor vadd_cuda_forward(torch::Tensor A, torch::Tensor B, int threads)
{
    auto C = torch::zeros_like(A);

    const int blocks = (A.size(0) + threads - 1) / threads;

    AT_DISPATCH_FLOATING_TYPES(A.type(), "vadd_forward_cuda", ([&]
                                                               { vadd_cuda_forward_kernel<scalar_t><<<blocks, threads>>>(
                                                                     A.packed_accessor<scalar_t, 1, torch::RestrictPtrTraits, size_t>(),
                                                                     B.packed_accessor<scalar_t, 1, torch::RestrictPtrTraits, size_t>(),
                                                                     C.packed_accessor<scalar_t, 1, torch::RestrictPtrTraits, size_t>()); }));

    return C;
}