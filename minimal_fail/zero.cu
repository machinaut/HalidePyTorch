#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

namespace
{
    __global__ void _zero(float *A, int n)
    {
        printf("Hello from block %d, thread %d\n", blockIdx.x, threadIdx.x);
        int i = blockDim.x * blockIdx.x + threadIdx.x;
        if (i < n)
        {
            printf("Writing to i=%d.\n", i);
            printf("A[i] = %f.\n", A[i]);
            A[i] = 0;
        }
        else
        {
            printf("Skipping index %d\n", i);
        }
    }
}

extern "C" void zero(float *A, int n, int threads)
{
    printf("Got A pointer %p n %d threads %d\n", A, n, threads);
    const char *res_str;
    hipDeviceptr_t pbase;
    size_t psize;
    hipError_t res = hipMemGetAddressRange(&pbase, &psize, (hipDeviceptr_t)A);
    hipDrvGetErrorName(res, &res_str);
    if (res != hipSuccess)
        printf("Error getting A mem address range: (%d) %s\n", res, res_str);
    else
        printf("A mem address range: 0x%p - 0x%p, size: %zu\n", (void *)pbase, (void *)pbase + psize, psize);

    printf("Running the kernel\n");
    const int blocks = (n + threads - 1) / threads;
    _zero<<<blocks, threads>>>(A, n);
    printf("Finished running the kernel\n");

    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
        printf("Failed to launch zero kernel (error code %s)!\n", hipGetErrorString(err));
    else
        printf("Successfully launched zero kernel!\n");
}
